#include "hip/hip_runtime.h"
/*
	CUDA implementation of the Bellman-Ford's algorithm.

	Version BF0-none-AoS-noSh:
	- the input graph is stored as an array of weighted arcs (Array of Structures),
	- the parallelization is done on the "inner cycle",
	- no mutexes
	- no shared memory
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "utils.h"

// CUDA block's size for monodimensional grid
#define BLKDIM 1024

/*
	CUDA kernel of Bellman-Ford's algorithm.
	Each thread executes a relax on a single edge in each kernel call.
*/
__global__ void cuda_bellman_ford(uint32_t n_edges, Edge *graph, uint32_t *distances) {
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n_edges) {
		// relax the edge (u,v)
		const uint32_t u = graph[idx].start_node;
		const uint32_t v = graph[idx].end_node;
		// overflow-safe check
		if (distances[v] > distances[u] && distances[v] - distances[u] > graph[idx].weight) {
			distances[v] = distances[u] + graph[idx].weight;
		}
	}
}

/*
	Executes the Bellman-Ford's algorithm on the graph |h_graph|.
	Returns a pointer to an array with |n_nodes| elements:
	each element of index |i| contains the shortest path distance from node
	|source| to node |i|.
*/
uint32_t *bellman_ford(Edge *h_graph, uint32_t n_nodes, uint32_t n_edges, uint32_t source) {
	if (h_graph == NULL) {
		return NULL;
	}

	if (source >= n_nodes) {
		fprintf(stderr, "ERROR: source node %u does not exist\n\n", source);
		exit(EXIT_FAILURE);
	}

	size_t sz_distances = n_nodes * sizeof(uint32_t);
	size_t sz_graph = n_edges * sizeof(Edge);

	Edge *d_graph;

	uint32_t *d_distances;
	uint32_t *h_distances = (uint32_t *)malloc(sz_distances);
	assert(h_distances);

	for (uint32_t i = 0; i < n_nodes; i++) {
		h_distances[i] = UINT_MAX;
	}
	h_distances[source] = 0;

	// malloc and copy of the distances array
	cudaSafeCall(hipMalloc((void **)&d_distances, sz_distances));
	cudaSafeCall(hipMemcpy(d_distances, h_distances, sz_distances, hipMemcpyHostToDevice));

	// malloc and copy of the graph
	cudaSafeCall(hipMalloc((void **)&d_graph, sz_graph));
	cudaSafeCall(hipMemcpy(d_graph, h_graph, sz_graph, hipMemcpyHostToDevice));

	for (uint32_t i = 0; i < n_nodes - 1; i++) {
		cuda_bellman_ford<<<(n_edges + BLKDIM - 1) / BLKDIM, BLKDIM>>>(n_edges, d_graph, d_distances);
		cudaCheckError();
	}

	// copy-back of the result
	cudaSafeCall(hipMemcpy(h_distances, d_distances, sz_distances, hipMemcpyDeviceToHost));

	// deallocation
	hipFree(d_graph);
	hipFree(d_distances);

	return h_distances;
}

int main(int argc, char *argv[]) {
	if (argc < 2 || argc > 3) {
		fprintf(stderr, "Usage: %s <input_file> [<solution_file>]\n\n", argv[0]);
		return EXIT_FAILURE;
	}

	Edge *graph;
	uint32_t nodes;
	uint32_t edges;
	uint32_t *result;

	clock_t compute_start;
	clock_t compute_end;

	fprintf(stderr, "Reading input graph...");
	graph = read_graph(argv[1], &nodes, &edges);
	fprintf(stderr, "done\n");

	fprintf(stderr, "\nGraph data:\n");
	fprintf(stderr, " %7u nodes\n", nodes);
	fprintf(stderr, " %7u arcs\n", edges);

	print_ram_usage(sizeof(Edge) * edges);

	fprintf(stderr, "Computing Bellman-Ford...");
	compute_start = clock();
	result = bellman_ford(graph, nodes, edges, 0);
	compute_end = clock();
	fprintf(stderr, "done\n\n");

	const float compute_seconds = (float)(compute_end - compute_start) / (float)CLOCKS_PER_SEC;
	fprintf(stderr, "\nActual execution time: %.3f seconds\n", compute_seconds);

	uint64_t total_work = (uint64_t)nodes * (uint64_t)edges;
	double throughput = (double)total_work / (double)compute_seconds;
	fprintf(stderr, "\nThroughput: %.3e relax/second\n\n", throughput);

	if (argc == 3) {
		uint32_t *distances = (uint32_t *)malloc(nodes * sizeof(uint32_t));

		fprintf(stderr, "Reading solution...");
		read_solution(argv[2], distances);
		fprintf(stderr, "done\n");

		check_solution(nodes, distances, result);

		free(distances);
	} else {
		fprintf(stderr, "Dumping solution...");
		dump_solution(nodes, 0, result);
		fprintf(stderr, "done\n");
	}

	free(graph);
	free(result);

	return EXIT_SUCCESS;
}