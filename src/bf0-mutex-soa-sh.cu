#include "hip/hip_runtime.h"
/*
    CUDA implementation of the Bellman-Ford's algorithm
    Copyright (C) 2021  Filippo Barbari

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/
/*
    CUDA implementation of the Bellman-Ford's algorithm.

    Version BF0-mutex-SoA-Sh:
    - the input graph is stored as an array of weighted arcs (Structure of Arrays),
    - the parallelization is done on the "inner cycle",
    - an atomic operation is used for the update of distances
    - a shared memory buffer is used

    To compile:
    nvcc -arch=<cuda_capability> bf0-mutex-aos.cu -o bf0-mutex-aos

    To run:
    ./bf0-mutex-aos < test/graph.txt > solution.txt
*/

#include "hpc.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA block's size for monodimensional grid
#define BLKDIM 1024

typedef struct {
    // The index of the source node of the edge
    unsigned int *start_nodes;

    // The index of the destination node of the edge
    unsigned int *end_nodes;

    // The weight assigned to the edge
    float *weights;
} Graph;

/*
    Reads a graph from stdin formatted as follows:
    first line: |number of nodes| |number of arcs| n
    each one of the other |number of arcs| lines: |source node| |destination node| |arc weight|

    The variables pointed by |n_nodes| and |n_edges| are modified accordingly.

    This function returns a pointer to a Graph structure.
*/
Graph* read_graph ( unsigned int *n_nodes, unsigned int *n_edges ) {
    /*
        |tmp| is necessary to read the third value of the first line, which is useless
    */
    unsigned int tmp;
    scanf("%u %u %u", n_nodes, n_edges, &tmp);

    Graph *graph = (Graph*) malloc(sizeof(Graph));
    assert(graph);

    graph->start_nodes = (unsigned int*) malloc((*n_edges) * sizeof(unsigned int));
    assert(graph->start_nodes);
    graph->end_nodes = (unsigned int*) malloc((*n_edges) * sizeof(unsigned int));
    assert(graph->end_nodes);
    graph->weights = (float*) malloc((*n_edges) * sizeof(float));
    assert(graph->weights);

    for(unsigned int i=0; i<*n_edges; i++) {
        scanf("%u %u %f", &graph->start_nodes[i], &graph->end_nodes[i], &graph->weights[i]);

        if(graph->start_nodes[i] >= *n_nodes || graph->end_nodes[i] >= *n_nodes) {
            fprintf(stderr, "ERROR at line %u: invalid node index.\n\n", i+1);
            exit(EXIT_FAILURE);
        }
    }

    return graph;
}

/*
    Dumps the solution on stdout.

    Output is formatted as follows:

    number_of_nodes
    source_node
    node_0 distance_to_node_0
    node_1 distance_to_node_1
    node_2 distance_to_node_2
    ...
*/
void dump_solution (unsigned int n_nodes, unsigned int source, float *dist) {
    printf("%u\n%u\n", n_nodes, source);

    for(unsigned int i=0; i<n_nodes; i++) {
        printf("%u", i);
        if(isinf(dist[i])) {
            printf(" %u\n", UINT_MAX);
        }
        else {
            printf(" %u\n", (unsigned int)dist[i]);
        }
    }
}

/*
    CUDA kernel of Bellman-Ford's algorithm.
    Each thread executes a relax on a single edge in each kernel call.
*/
__global__ void cuda_bellman_ford (unsigned int n_edges,
                                   unsigned int* start_nodes,
                                   unsigned int* end_nodes,
                                   float* weights,
                                   float *distances) {
    union {
        float vf;
        int vi;
    } oldval, newval;

    __shared__ unsigned int buffer[3 * BLKDIM];
    unsigned int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int l_idx = 3 * threadIdx.x;

    // Filling the shared memory buffer
    if(g_idx < n_edges) {
        buffer[l_idx]   = start_nodes[g_idx];
        buffer[l_idx+1] = end_nodes[g_idx];
        buffer[l_idx+2] = weights[g_idx];
    }
    __syncthreads();

    if(g_idx < n_edges) {
        // relax the edge (u,v)
        const unsigned int u = buffer[l_idx];
        const unsigned int v = buffer[l_idx+1];

        if(distances[u] + buffer[l_idx+2] < distances[v]) {
            do {
                oldval.vf = distances[v];
                newval.vf = distances[u] + buffer[l_idx+2];
                newval.vf = fminf(oldval.vf, newval.vf);
            } while( atomicCAS((int*)&distances[v], oldval.vi, newval.vi) != newval.vi );
        }
    }
}

/*
    Executes the Bellman-Ford's algorithm on the graph |h_graph|.
    Returns a pointer to an array with |n_nodes| elements:
    each element of index |i| contains the shortest path distance from node
    |source| to node |i|.
*/
float* bellman_ford ( Graph* h_graph, unsigned int n_nodes, unsigned int n_edges, unsigned int source ) {
    if(h_graph == NULL) return NULL;
    if(source >= n_nodes) {
        fprintf(stderr, "ERROR: source node %u does not exist\n\n", source);
        exit(EXIT_FAILURE);
    }

    size_t sz_distances = n_nodes * sizeof(float);
    size_t sz = n_edges * sizeof(unsigned int);

    unsigned int* d_start_nodes;
    unsigned int* d_end_nodes;
    float* d_weights;

    float *d_distances;
    float *h_distances = (float*) malloc(sz_distances);
    assert(h_distances);

    for(unsigned int i=0; i<n_nodes; i++) {
        h_distances[i] = HUGE_VAL;
    }
    h_distances[source] = 0.0f;

    // malloc and copy of the distances array
    cudaSafeCall( hipMalloc((void**)&d_distances, sz_distances) );
    cudaSafeCall( hipMemcpy(d_distances, h_distances, sz_distances, hipMemcpyHostToDevice) );

    // malloc and copy of the graph
    cudaSafeCall( hipMalloc((void**)&d_start_nodes, sz) );
    cudaSafeCall( hipMemcpy(d_start_nodes, h_graph->start_nodes, sz, hipMemcpyHostToDevice) );
    cudaSafeCall( hipMalloc((void**)&d_end_nodes, sz) );
    cudaSafeCall( hipMemcpy(d_end_nodes, h_graph->end_nodes, sz, hipMemcpyHostToDevice) );
    cudaSafeCall( hipMalloc((void**)&d_weights, sz) );
    cudaSafeCall( hipMemcpy(d_weights, h_graph->weights, sz, hipMemcpyHostToDevice) );

    for(unsigned int i=0; i<n_nodes-1; i++) {
        cuda_bellman_ford <<< (n_edges+BLKDIM-1) / BLKDIM, BLKDIM >>> (n_edges, d_start_nodes, d_end_nodes, d_weights, d_distances);
        cudaCheckError();
    }

    // copy-back of the result
    cudaSafeCall( hipMemcpy(h_distances, d_distances, sz_distances, hipMemcpyDeviceToHost) );

    // deallocation
    hipFree(d_start_nodes);
    hipFree(d_end_nodes);
    hipFree(d_weights);
    hipFree(d_distances);

    return h_distances;
}

int main ( void ) {

    Graph *graph;
    unsigned int nodes, edges;
    float *result;

    clock_t program_start, program_end, compute_start, compute_end;

    program_start = clock();

    fprintf(stderr, "Reading input graph...");
    graph = read_graph(&nodes, &edges);
    fprintf(stderr, "done\n");

    fprintf(stderr, "\nGraph data:\n");
    fprintf(stderr, " %7u nodes\n", nodes);
    fprintf(stderr, " %7u arcs\n", edges);

    float ram_usage = (float)(3 * edges * sizeof(unsigned int));
    if(ram_usage < 1024.0f) {
        fprintf(stderr, " %.3f bytes of RAM used\n\n", ram_usage);
    }
    else if(ram_usage < 1024.0f*1024.0f) {
        fprintf(stderr, " %.3f KBytes of RAM used\n\n", ram_usage/1024.0f);
    }
    else {
        fprintf(stderr, " %.3f MBytes of RAM used\n\n", ram_usage/(1024.0f*1024.0f));
    }

    fprintf(stderr, "Computing Bellman-Ford...");
    compute_start = clock();
    result = bellman_ford(graph, nodes, edges, 0);
    compute_end = clock();
    fprintf(stderr, "done\n\n");

    fprintf(stderr, "Dumping solution...");
    dump_solution(nodes, 0, result);
    fprintf(stderr, "done\n");

    free(graph);
    free(result);

    program_end = clock();

    float total_seconds = (float)(program_end-program_start) / (float)CLOCKS_PER_SEC;
    float compute_seconds = (float)(compute_end-compute_start) / (float)CLOCKS_PER_SEC;

    fprintf(stderr, "\nTotal execution time: %.3f seconds\n", total_seconds);
    fprintf(stderr, "Actual execution time: %.3f seconds\n", compute_seconds);

    unsigned long long total_work = (unsigned long long) nodes * (unsigned long long) edges;
    double throughput = (double)total_work / (double)compute_seconds;
    fprintf(stderr, "\nThroughput: %.3e relax/second\n", throughput);

    return EXIT_SUCCESS;
}