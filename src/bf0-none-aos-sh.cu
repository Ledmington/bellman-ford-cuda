#include "hip/hip_runtime.h"
/*
	CUDA implementation of the Bellman-Ford's algorithm.

	Version BF0-none-AoS-Sh:
	- the input graph is stored as an array of weighted arcs (Array of Structures),
	- the parallelization is done on the "inner cycle",
	- no mutexes
	- a shared memory buffer is used
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "utils.h"

// CUDA block's size for monodimensional grid
#define BLKDIM 1024

/*
	Reads a graph from stdin formatted as follows:
	first line: |number of nodes| |number of arcs| n
	each one of the other |number of arcs| lines: |source node| |destination
   node| |arc weight|

	The variables pointed by |n_nodes| and |n_edges| are modified accordingly.

	This function returns a pointer to an array of |n_edges| structures of type
   Edge.
*/
Edge *read_graph(uint32_t *n_nodes, uint32_t *n_edges) {
	/*
		|tmp| is necessary to read the third value of the first line, which is
	   useless
	*/
	uint32_t tmp;
	scanf("%u %u %u", n_nodes, n_edges, &tmp);

	Edge *graph = (Edge *)malloc((*n_edges) * sizeof(Edge));
	assert(graph);

	for (uint32_t i = 0; i < *n_edges; i++) {
		float tmp;
		scanf("%u %u %f", &graph[i].start_node, &graph[i].end_node, &tmp);
		graph[i].weight = (uint32_t)tmp;

		if (graph[i].start_node >= *n_nodes || graph[i].end_node >= *n_nodes) {
			fprintf(stderr, "ERROR at line %u: invalid node index\n\n", i + 1);
			exit(EXIT_FAILURE);
		}
	}

	return graph;
}

/*
	CUDA kernel of Bellman-Ford's algorithm.
	Each thread executes a relax on a single edge in each kernel call.
*/
__global__ void cuda_bellman_ford(uint32_t n_edges, Edge *graph, uint32_t *distances) {
	__shared__ Edge buffer[BLKDIM];
	uint32_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t l_idx = threadIdx.x;

	// Filling the shared memory buffer
	if (g_idx < n_edges) {
		buffer[l_idx] = graph[g_idx];
	}
	__syncthreads();

	if (g_idx < n_edges) {
		// relax the edge (u,v)
		const uint32_t u = buffer[l_idx].start_node;
		const uint32_t v = buffer[l_idx].end_node;
		// overflow-safe check
		if (distances[v] > distances[u] && distances[v] - distances[u] > buffer[l_idx].weight) {
			distances[v] = distances[u] + buffer[l_idx].weight;
		}
	}
}

/*
	Executes the Bellman-Ford's algorithm on the graph |h_graph|.
	Returns a pointer to an array with |n_nodes| elements:
	each element of index |i| contains the shortest path distance from node
	|source| to node |i|.
*/
uint32_t *bellman_ford(Edge *h_graph, uint32_t n_nodes, uint32_t n_edges, uint32_t source) {
	if (h_graph == NULL)
		return NULL;
	if (source >= n_nodes) {
		fprintf(stderr, "ERROR: source node %u does not exist\n\n", source);
		exit(EXIT_FAILURE);
	}

	size_t sz_distances = n_nodes * sizeof(uint32_t);
	size_t sz_graph = n_edges * sizeof(Edge);

	Edge *d_graph;

	uint32_t *d_distances;
	uint32_t *h_distances = (uint32_t *)malloc(sz_distances);
	assert(h_distances);

	for (uint32_t i = 0; i < n_nodes; i++) {
		h_distances[i] = UINT_MAX;
	}
	h_distances[source] = 0;

	// malloc and copy of the distances array
	cudaSafeCall(hipMalloc((void **)&d_distances, sz_distances));
	cudaSafeCall(hipMemcpy(d_distances, h_distances, sz_distances, hipMemcpyHostToDevice));

	// malloc and copy of the graph
	cudaSafeCall(hipMalloc((void **)&d_graph, sz_graph));
	cudaSafeCall(hipMemcpy(d_graph, h_graph, sz_graph, hipMemcpyHostToDevice));

	for (uint32_t i = 0; i < n_nodes - 1; i++) {
		cuda_bellman_ford<<<(n_edges + BLKDIM - 1) / BLKDIM, BLKDIM>>>(n_edges, d_graph, d_distances);
		cudaCheckError();
	}

	// copy-back of the result
	cudaSafeCall(hipMemcpy(h_distances, d_distances, sz_distances, hipMemcpyDeviceToHost));

	// deallocation
	hipFree(d_graph);
	hipFree(d_distances);

	return h_distances;
}

int main(void) {
	Edge *graph;
	uint32_t nodes, edges;
	uint32_t *result;

	clock_t program_start, program_end, compute_start, compute_end;

	program_start = clock();

	fprintf(stderr, "Reading input graph...");
	graph = read_graph(&nodes, &edges);
	fprintf(stderr, "done\n");

	fprintf(stderr, "\nGraph data:\n");
	fprintf(stderr, " %7u nodes\n", nodes);
	fprintf(stderr, " %7u arcs\n", edges);

	print_ram_usage(sizeof(Edge) * edges);

	fprintf(stderr, "Computing Bellman-Ford...");
	compute_start = clock();
	result = bellman_ford(graph, nodes, edges, 0);
	compute_end = clock();
	fprintf(stderr, "done\n\n");

	fprintf(stderr, "Dumping solution...");
	dump_solution(nodes, 0, result);
	fprintf(stderr, "done\n");

	free(graph);
	free(result);

	program_end = clock();

	float total_seconds = (float)(program_end - program_start) / (float)CLOCKS_PER_SEC;
	float compute_seconds = (float)(compute_end - compute_start) / (float)CLOCKS_PER_SEC;

	fprintf(stderr, "\nTotal execution time: %.3f seconds\n", total_seconds);
	fprintf(stderr, "Actual execution time: %.3f seconds\n", compute_seconds);

	unsigned long long total_work = (unsigned long long)nodes * (unsigned long long)edges;
	double throughput = (double)total_work / (double)compute_seconds;
	fprintf(stderr, "\nThroughput: %.3e relax/second\n", throughput);

	return EXIT_SUCCESS;
}