#include "hip/hip_runtime.h"
/*
	CUDA implementation of the Bellman-Ford's algorithm.

	Version BF0-none-AoS-Sh:
	- the input graph is stored as an array of weighted arcs (Array of Structures),
	- the parallelization is done on the "inner cycle",
	- no mutexes
	- a shared memory buffer is used
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "utils.h"

// CUDA block's size for monodimensional grid
#define BLKDIM 1024

/*
	CUDA kernel of Bellman-Ford's algorithm.
	Each thread executes a relax on a single edge in each kernel call.
*/
__global__ void cuda_bellman_ford(uint32_t n_edges, Edge *graph, uint32_t *distances) {
	__shared__ Edge buffer[BLKDIM];
	uint32_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t l_idx = threadIdx.x;

	// Filling the shared memory buffer
	if (g_idx < n_edges) {
		buffer[l_idx] = graph[g_idx];
	}
	__syncthreads();

	if (g_idx < n_edges) {
		// relax the edge (u,v)
		const uint32_t u = buffer[l_idx].start_node;
		const uint32_t v = buffer[l_idx].end_node;
		// overflow-safe check
		if (distances[v] > distances[u] && distances[v] - distances[u] > buffer[l_idx].weight) {
			distances[v] = distances[u] + buffer[l_idx].weight;
		}
	}
}

/*
	Executes the Bellman-Ford's algorithm on the graph |h_graph|.
	Returns a pointer to an array with |n_nodes| elements:
	each element of index |i| contains the shortest path distance from node
	|source| to node |i|.
*/
uint32_t *bellman_ford(Edge *h_graph, uint32_t n_nodes, uint32_t n_edges, uint32_t source) {
	if (h_graph == NULL)
		return NULL;
	if (source >= n_nodes) {
		fprintf(stderr, "ERROR: source node %u does not exist\n\n", source);
		exit(EXIT_FAILURE);
	}

	size_t sz_distances = n_nodes * sizeof(uint32_t);
	size_t sz_graph = n_edges * sizeof(Edge);

	Edge *d_graph;

	uint32_t *d_distances;
	uint32_t *h_distances = (uint32_t *)malloc(sz_distances);
	assert(h_distances);

	for (uint32_t i = 0; i < n_nodes; i++) {
		h_distances[i] = UINT_MAX;
	}
	h_distances[source] = 0;

	// malloc and copy of the distances array
	cudaSafeCall(hipMalloc((void **)&d_distances, sz_distances));
	cudaSafeCall(hipMemcpy(d_distances, h_distances, sz_distances, hipMemcpyHostToDevice));

	// malloc and copy of the graph
	cudaSafeCall(hipMalloc((void **)&d_graph, sz_graph));
	cudaSafeCall(hipMemcpy(d_graph, h_graph, sz_graph, hipMemcpyHostToDevice));

	for (uint32_t i = 0; i < n_nodes - 1; i++) {
		cuda_bellman_ford<<<(n_edges + BLKDIM - 1) / BLKDIM, BLKDIM>>>(n_edges, d_graph, d_distances);
		cudaCheckError();
	}

	// copy-back of the result
	cudaSafeCall(hipMemcpy(h_distances, d_distances, sz_distances, hipMemcpyDeviceToHost));

	// deallocation
	hipFree(d_graph);
	hipFree(d_distances);

	return h_distances;
}

int main(void) {
	Edge *graph;
	uint32_t nodes, edges;
	uint32_t *result;

	clock_t program_start, program_end, compute_start, compute_end;

	program_start = clock();

	fprintf(stderr, "Reading input graph...");
	graph = read_graph(&nodes, &edges);
	fprintf(stderr, "done\n");

	fprintf(stderr, "\nGraph data:\n");
	fprintf(stderr, " %7u nodes\n", nodes);
	fprintf(stderr, " %7u arcs\n", edges);

	print_ram_usage(sizeof(Edge) * edges);

	fprintf(stderr, "Computing Bellman-Ford...");
	compute_start = clock();
	result = bellman_ford(graph, nodes, edges, 0);
	compute_end = clock();
	fprintf(stderr, "done\n\n");

	fprintf(stderr, "Dumping solution...");
	dump_solution(nodes, 0, result);
	fprintf(stderr, "done\n");

	free(graph);
	free(result);

	program_end = clock();

	float total_seconds = (float)(program_end - program_start) / (float)CLOCKS_PER_SEC;
	float compute_seconds = (float)(compute_end - compute_start) / (float)CLOCKS_PER_SEC;

	fprintf(stderr, "\nTotal execution time: %.3f seconds\n", total_seconds);
	fprintf(stderr, "Actual execution time: %.3f seconds\n", compute_seconds);

	unsigned long long total_work = (unsigned long long)nodes * (unsigned long long)edges;
	double throughput = (double)total_work / (double)compute_seconds;
	fprintf(stderr, "\nThroughput: %.3e relax/second\n", throughput);

	return EXIT_SUCCESS;
}