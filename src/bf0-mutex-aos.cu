#include "hip/hip_runtime.h"
/*
    CUDA implementation of the Bellman-Ford's algorithm
    Copyright (C) 2021  Filippo Barbari

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/
/*
    CUDA implementation of the Bellman-Ford's algorithm.

    Version BF0-mutex-AoS:
    - the input graph is stored as an array of weighted arcs (Array of Structures),
    - the parallelization is done on the "inner cycle",
    - an atomic operation is used for the update of distances

    To compile:
    nvcc -arch=<cuda_capability> bf0-mutex-aos.cu -o bf0-mutex-aos

    To run:
    ./bf0-mutex-aos < test/graph.txt > solution.txt
*/

#include "hpc.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA block's size for monodimensional grid
#define BLKDIM 1024

typedef struct {
    // The index of the source node of the edge
    unsigned int start_node;

    // The index of the destination node of the edge
    unsigned int end_node;

    // The weight assigned to the edge
    float weight;
} Edge;

/*
    Reads a graph from stdin formatted as follows:
    first line: |number of nodes| |number of arcs| n
    each one of the other |number of arcs| lines: |source node| |destination node| |arc weight|

    The variables pointed by |n_nodes| and |n_edges| are modified accordingly.

    This function returns a pointer to an array of |n_edges| structures of type Edge.
*/
Edge* read_graph ( unsigned int *n_nodes, unsigned int *n_edges ) {
    /*
        |tmp| is necessary to read the third value of the first line, which is useless
    */
    unsigned int tmp;
    scanf("%u %u %u", n_nodes, n_edges, &tmp);

    Edge *graph = (Edge*) malloc(*n_edges * sizeof(Edge));
    assert(graph);

    for(unsigned int i=0; i<*n_edges; i++) {
        scanf("%u %u %f", &graph[i].start_node, &graph[i].end_node, &graph[i].weight);

        if(graph[i].start_node >= *n_nodes || graph[i].end_node >= *n_nodes) {
            fprintf(stderr, "ERROR at line %u: invalid node index.\n\n", i+1);
            exit(EXIT_FAILURE);
        }
    }

    return graph;
}

/*
    Dumps the solution on stdout.

    Output is formatted as follows:

    number_of_nodes
    source_node
    node_0 distance_to_node_0
    node_1 distance_to_node_1
    node_2 distance_to_node_2
    ...
*/
void dump_solution (unsigned int n_nodes, unsigned int source, float *dist) {
    printf("%u\n%u\n", n_nodes, source);

    for(unsigned int i=0; i<n_nodes; i++) {
        printf("%u", i);
        if(isinf(dist[i])) {
            printf(" %u\n", UINT_MAX);
        }
        else {
            printf(" %u\n", (unsigned int)dist[i]);
        }
    }
}

/*
    CUDA kernel of Bellman-Ford's algorithm.
    Each thread executes a relax on a single edge in each kernel call.
*/
__global__ void cuda_bellman_ford (unsigned int n_edges,
                                   Edge* graph,
                                   float *distances) {
    union {
        float vf;
        int vi;
    } oldval, newval;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n_edges) {
        // relax the edge (u,v)
        const unsigned int u = graph[idx].start_node;
        const unsigned int v = graph[idx].end_node;

        if(distances[u] + graph[idx].weight < distances[v]) {
            do {
                oldval.vf = distances[v];
                newval.vf = distances[u] + graph[idx].weight;
                newval.vf = fminf(oldval.vf, newval.vf);
            } while( atomicCAS((int*)&distances[v], oldval.vi, newval.vi) != newval.vi );
        }
    }
}

/*
    Executes the Bellman-Ford's algorithm on the graph |h_graph|.
    Returns a pointer to an array with |n_nodes| elements:
    each element of index |i| contains the shortest path distance from node
    |source| to node |i|.
*/
float* bellman_ford ( Edge* h_graph, unsigned int n_nodes, unsigned int n_edges, unsigned int source ) {
    if(h_graph == NULL) return NULL;
    if(source >= n_nodes) {
        fprintf(stderr, "ERROR: source node %u does not exist\n\n", source);
        exit(EXIT_FAILURE);
    }

    size_t sz_distances = n_nodes * sizeof(float);
    size_t sz_graph = n_edges * sizeof(Edge);

    Edge* d_graph;

    float *d_distances;
    float *h_distances = (float*) malloc(sz_distances);
    assert(h_distances);

    for(unsigned int i=0; i<n_nodes; i++) {
        h_distances[i] = HUGE_VAL;
    }
    h_distances[source] = 0.0f;

    // malloc and copy of the distances array
    cudaSafeCall( hipMalloc((void**)&d_distances, sz_distances) );
    cudaSafeCall( hipMemcpy(d_distances, h_distances, sz_distances, hipMemcpyHostToDevice) );

    // malloc and copy of the graph
    cudaSafeCall( hipMalloc((void**)&d_graph, sz_graph) );
    cudaSafeCall( hipMemcpy(d_graph, h_graph, sz_graph, hipMemcpyHostToDevice) );

    for(unsigned int i=0; i<n_nodes-1; i++) {
        cuda_bellman_ford <<< (n_edges+BLKDIM-1) / BLKDIM, BLKDIM >>> (n_edges, d_graph, d_distances);
        cudaCheckError();
    }

    // copy-back of the result
    cudaSafeCall( hipMemcpy(h_distances, d_distances, sz_distances, hipMemcpyDeviceToHost) );

    // deallocation
    hipFree(d_graph);
    hipFree(d_distances);

    return h_distances;
}

int main ( void ) {

    Edge *graph;
    unsigned int nodes, edges;
    float *result;

    clock_t program_start, program_end, compute_start, compute_end;

    program_start = clock();

    fprintf(stderr, "Reading input graph...");
    graph = read_graph(&nodes, &edges);
    fprintf(stderr, "done\n");

    fprintf(stderr, "\nGraph data:\n");
    fprintf(stderr, " %7u nodes\n", nodes);
    fprintf(stderr, " %7u arcs\n", edges);

    float ram_usage = (float)(sizeof(Edge)*edges);
    if(ram_usage < 1024.0f) {
        fprintf(stderr, " %.3f bytes of RAM used\n\n", ram_usage);
    }
    else if(ram_usage < 1024.0f*1024.0f) {
        fprintf(stderr, " %.3f KBytes of RAM used\n\n", ram_usage/1024.0f);
    }
    else {
        fprintf(stderr, " %.3f MBytes of RAM used\n\n", ram_usage/(1024.0f*1024.0f));
    }

    fprintf(stderr, "Computing Bellman-Ford...");
    compute_start = clock();
    result = bellman_ford(graph, nodes, edges, 0);
    compute_end = clock();
    fprintf(stderr, "done\n\n");

    fprintf(stderr, "Dumping solution...");
    dump_solution(nodes, 0, result);
    fprintf(stderr, "done\n");

    free(graph);
    free(result);

    program_end = clock();

    fprintf(stderr, "\nTotal execution time: %.3f seconds\n", (float)(program_end-program_start) / (float)CLOCKS_PER_SEC);
    fprintf(stderr, "Actual execution time: %.3f seconds\n", (float)(compute_end-compute_start) / (float)CLOCKS_PER_SEC);

    return EXIT_SUCCESS;
}